#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <numeric>
#include <string>
#include <functional>

using namespace std;

#define MAX_BLOCK_SIZE 1024

__device__ bool found;

hipError_t customCudaError(hipError_t result)
{
	if (result != hipSuccess)
    	{
        	fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
	        assert(result == hipSuccess);
    	}
   	return result;
}

__device__
void displayTable(unsigned int *plain, unsigned int *hash, unsigned int columnCount, unsigned int rowCount)
{
	printf("Rainbow table (row=%d x depth=%d) :\n", rowCount, columnCount);
	for(int i = 0; i < rowCount; i++)
	{
		printf("PLAIN : %d | HASH : %d\n", plain[i], hash[i]);
	}
}

__device__
void hashingKernel(unsigned int plain, unsigned int *hash)
{
	// Hashing kernel (36669 => 174576660)
	*hash = ((plain >> 16) ^ plain) * 0x45;
	*hash = ((*hash >> 16) ^ *hash) * 0x45;
	*hash = (*hash >> 16) ^ *hash;
}

__device__
void reductionKernel(unsigned int maxValue, unsigned int hash, unsigned int *reduction)
{
	// Reduction kernel (174576660 => 17457)
	while (hash > maxValue)
	{
		hash = hash / 10;
	}
	*reduction = hash;

}

__global__
void findingKernel(unsigned int *plainArray, unsigned int *hashArray, unsigned int hash, unsigned int columnCount, unsigned int rowCount, unsigned int maxValue)
{
	int th = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int localHash = hashArray[th];
	unsigned int plain = plainArray[th];
	unsigned int reduction;

	while (!found)
	{
		if (localHash == hash)
		{
			for (int i = 0; i < columnCount; i++)
			{
				hashingKernel(plain, &localHash);
				if (localHash == hash)
				{
					printf("#### Match for %d (HASH : %d) on Thread %d ####\n", plain, localHash, th);
					found = true;
					__threadfence();
					break;
				}
				else
				{
					reduction = localHash;
					reductionKernel(maxValue, localHash, &reduction);
					plain = reduction;
					__syncthreads();
				}
			}

		}
		else
		{
			reductionKernel(maxValue, localHash, &reduction);
			plain = reduction;
			hashingKernel(plain, &localHash);
			reduction = localHash;
			__syncthreads();
		}
	}

}


void rainbowWrapper(unsigned int rowCount, unsigned int columnCount, unsigned int maxValue, unsigned int *plainArray, unsigned int *hashArray, boolean display)
{
	unsigned int *d_plainArray = NULL;
	unsigned int *d_hashArray = NULL;
	hipEvent_t start;
	hipEvent_t stop;

	// Allocate memory space on the device
	customCudaError(hipMalloc((void **)&d_plainArray, sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE));
	customCudaError(hipMalloc((void **)&d_hashArray, sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE));

	// Copy data on the device
	customCudaError(hipMemcpy(d_plainArray, plainArray, sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE, hipMemcpyHostToDevice));

	// Lauch the rainbow table generation kernel
	rainbowKernel<<<rowCount,1024>>>(d_plainArray, d_hashArray, columnCount, maxValue);
	customCudaError(hipDeviceSynchronize());

	// Fetch the data from the device
	customCudaError(hipMemcpy(hashArray, d_hashArray, sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE, hipMemcpyDeviceToHost));

	// Record the start event for the second kernel
	customCudaError(hipEventCreate(&start));
	customCudaError(hipEventCreate(&stop));
	customCudaError(hipEventRecord(start, NULL));

	// Launch the hash resolver kernel
	printf("Searching for the hash into the table...\n");
	findingKernel<<<rowCount,1024>>>(d_plainArray, d_hashArray, hash, columnCount, rowCount, maxValue);
	customCudaError(hipDeviceSynchronize());

	// Display the table (or not)
	if (display == 1)
	{
		displayTable(plainArray, hashArray, columnCount, rowCount);
	}

	// Record the stop event for the first event
	customCudaError(hipEventRecord(stop, NULL));
	customCudaError(hipEventSynchronize(stop));

	// Display the time enlapsed informations
	printf("################\n");
	float msecTotal = 0.0f;
	customCudaError(hipEventElapsedTime(&msecTotal, start, stop));
	double gigaFlops = (columnCount * rowCount * MAX_BLOCK_SIZE * 1.0e-9f) / (msecTotal / 1000.0f);
	printf("Cuda processing time = %.3fms, Performance = %.3f GFlop/s\n", msecTotal, gigaFlops);

	customCudaError(hipFree(d_plainArray));
	customCudaError(hipFree(d_hashArray));
}

__global__
void rainbowKernel(unsigned int *plainArray, unsigned int *hashArray, unsigned int columnCount, unsigned int maxValue) {
	int th = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int plain = plainArray[th];
	unsigned int hash;
	unsigned int reduction;

	for (int i = 0; i < columnCount; i++)
	{
		hashingKernel(plain, &hash);
		reduction = hash;
		reductionKernel(maxValue, hash, &reduction);
		plain = reduction;
	}
	hashArray[th] = hash;
}

int main(int argc, char** argv) {

	unsigned int maxValue = 99999;
	unsigned int minValue = 11111;
	unsigned int rowCount = 4;
	unsigned int columnCount = 4096;
	boolean display = false;
	char * s_hash;
	unsigned int hash = 0;
 	unsigned int *plainArray = NULL;
	unsigned int *hashArray = NULL;

	// Clear the terminal
	system("clear");

	// Get the device
	int dev = findCudaDevice(argc, (const char **)argv);

	// Display the help
	if (checkCmdLineFlag(argc, (const char **)argv, "help") || checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
       	printf("Usage :\n");
		printf("      -hash=HASH [0] (Password hash you want to crack) \n");
		printf("      -block=BLOCK [4] (Rainbow table's row count (1 block = 1024 row)) \n");
		printf("      -depth=DEPTH [4096] (Rainbow table's column count) \n");
       	printf("      -verbose (Display the rainbow table)\n");

       	exit(EXIT_SUCCESS);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "hash"))
    {
       	getCmdLineArgumentString(argc, (const char **)argv, "hash", &s_hash);
       	hash = atoi(s_hash);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "row"))
    {
		rowCount = getCmdLineArgumentInt(argc, (const char**)argv, "row");
	}
	if (checkCmdLineFlag(argc, (const char **)argv, "depth"))
	{
		columnCount = getCmdLineArgumentInt(argc, (const char**)argv, "depth");
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
	{
		display = true;
	}

	plainArray = (unsigned int *)malloc(sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE);
	hashArray = (unsigned int *)malloc(sizeof(unsigned int) * rowCount * MAX_BLOCK_SIZE);

	printf("Generating random passwords...\n");
    srand(time(NULL));
    for (int i = 0; i < rowCount * MAX_BLOCK_SIZE; i++)
  	{
		plainArray[i] = rand() % (maxValue-minValue + 1) + minValue;
	}

	printf("Generation done\n");

	rainbowWrapper(rowCount, columnCount, maxValue, plainArray, hashArray, display)

	free(plainArray);
	free(hashArray);
    exit(EXIT_SUCCESS);

}
